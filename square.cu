// Square numbers parallely on GPU


#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel function which executes on GPU
__global__ void square(float *d_out, float *d_in)
{
   int idx = threadIdx.x;
   float f = d_in[idx];
   d_out[idx] = f * f;
}

int main()
{
	const int ARRAY_SIZE = 1024;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	float h_in[ARRAY_SIZE];
	for(int i = 0; i < ARRAY_SIZE; i++){
		h_in[i] = float(i);
	}
	
	float h_out[ARRAY_SIZE];

	float *d_in;
	float *d_out;

	//allocate memory on GPU device
	hipMalloc((void **) &d_in, ARRAY_BYTES);
	hipMalloc((void **) &d_out, ARRAY_BYTES);

	//copy input data in h_in from host to GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	//execute function in GPU
	square<<< 1, ARRAY_SIZE>>>(d_out, d_in);

	//Copy result back to host from device into h_out array
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	for(int i = 0; i < ARRAY_SIZE; i++){
		printf("%f", h_out[i]);
	    printf(((i % 4) != 3) ? "\t" : "\n");
	}

	//Free allocated memory by GPU
	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
