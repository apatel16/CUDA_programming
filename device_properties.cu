
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(){
    void printCudaDeviceProperties(void);

    printCudaDeviceProperties();
    return 0;
}

//ToDo - Get the information about gpu device
void printCudaDeviceProperties(){

    //function declaration
    int ConvertSMVersionNumberToCores(int, int);

    printf("Cuda Information :\n");
    printf("==============================================================\n");

    hipError_t ret_cuda_rt;
    int dev_count;
    
    ret_cuda_rt = hipGetDeviceCount(&dev_count);
    if(ret_cuda_rt != hipSuccess){
        printf("Cuda runtime API Error - cudaGetDeviceCount() failed due to %s. Exitting Now ...\n", hipGetErrorString(ret_cuda_rt));
    }
    else if(dev_count == 0){
        printf("No CUDA supported device on system.\n");
        return ;
    }
    else{
        printf("Total number of cuda supporting GPU devices on the system : %d\n", dev_count);
        
        // Print properties of each device
        for (int i = 0; i < dev_count; i++){
            hipDeviceProp_t dev_prop;
            int driverVersion = 0; 
            int runtimeVersion = 0;

            ret_cuda_rt = hipGetDeviceProperties(&dev_prop, i);
            if(ret_cuda_rt != hipSuccess){
                printf("%s in %s at line %d\n", hipGetErrorString(ret_cuda_rt), __FILE__, __LINE__);
                return;
            }

            printf("\n");
            hipDriverGetVersion(&driverVersion);
            hipRuntimeGetVersion(&runtimeVersion);

            printf("CUDA DRIVER AND RUNTIME INFORMATIO\n");
            printf("=================================================\n");
            printf("Cuda driver  version     : %d.%d\n", driverVersion/1000, (driverVersion % 100)/10);
            printf("Cuda runtime version     : %d.%d\n", runtimeVersion/1000, (runtimeVersion % 100)/10);

        }
    }


}

int ConvertSMVersionNumberToCores(int major, int minor){

    return 0;
}