
#include <hip/hip_runtime.h>
#include<stdio.h>

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__ void hello(){
    printf("Hello World! I am thread in block %d\n", blockIdx.x);
}

int main(int argc, char** argv){

    //launch a kernel
    hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

    //force to printf()s to flush()
    hipDeviceSynchronize();
    printf("END!!!!!\n");

    return 0;
}