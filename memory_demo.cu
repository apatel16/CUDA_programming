#include<stdio.h>


//
// Todo - Write Kernels

//


int main(int argc, char **argv){

    use_local_memory_GPU<<<1, 128>>>(2.0f);


    // array on host 
    float h_arr[128];

    // pointer for global memory on GPU
    float *d_arr;

    //allocate global memory on GPU device as store results in d_arr
    hipMalloc((void **)&d_arr, sizeof(float) * 128);

    //copy data from host memory to device memory
    hipMemcpy((void *)d_arr, (void *)h_arr, sizeof(float) * 128, hipMemcpyHostToDevice);

    //lauch the kernel
    use_global_memory_GPU<<<1, 128>>>(d_arr);

    //copy data from device memory to host memory
    hipMemcpy((void *)h_arr, (void *)d_arr, sizeof(float) * 128, hipMemcpyDeviceToHost);

    hipFree(d_arr);


    return 0;
}